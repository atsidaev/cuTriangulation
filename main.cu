#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/glu.h>
#include <GL/glut.h>
//#include <cutil_inline.h>
#include <iomanip>
#include <stdlib.h>
#include <stdio.h>
#include < math.h > 
#include <malloc.h>
#define SIZE 50
typedef struct point 
{
    float x;
    float y;
} Point;

typedef struct line 
{
    Point a;
    Point b;
} Line;
__device__ __host__ void quickSort(Point *mas, int size,int sortType,int t)
{
	int i = 1;
    int j = 2;
	
	while (i<size+1)
	{
		bool m=false;
		if(sortType==1)
		{
			if(mas[i-1].x==mas[i].x)
				m=mas[i-1].y<mas[i].y;
			else if(mas[i-1].y==mas[i].y)
			{
				m=t*mas[i-1].x>t*mas[i].x;
			}
			if ((m)||(mas[i-1].y<mas[i].y)) { i=j; j++; }
			else
			{
				Point tmp=mas[i];
				mas[i]=mas[i-1];
				mas[i-1]=tmp;
				i--;
				if (i==0) { i=j; j++; }
			}
		}
		else
		{
			if(mas[i-1].x==mas[i].x)
					m=mas[i-1].y>mas[i].y;
			if ((m)||(mas[i-1].x<mas[i].x)) { i=j; j++; }
			else
			{
				Point tmp=mas[i];
				mas[i]=mas[i-1];
				mas[i-1]=tmp;
			i--;
			if (i==0) { i=j; j++; }
			}
			
		}
	}
}
__device__  __host__ bool isIntersect(Point ax,Point bx,Point cx,Point dx)
 {
	float x1,x2,x3,x4;
	float y1,y2,y3,y4;
	float k1,k2;
	float b1,b2;
	float x,y;
	x1=ax.x;
	x2=bx.x;
	y1=ax.y;
	y2=bx.y;

	x3=cx.x;
	x4=dx.x;
	y3=cx.y;
	y4=dx.y;


	if(y1==y2||x1==x2)k1=0;
	else k1=(y2-y1)/(x2-x1);
	if(y3==y4||x3==x4)k2=0;
	else k2=(y4-y3)/(x4-x3);
	if(k1==k2&&k1!=0)return false;
	b1=y1-k1*x1;
	b2=y3-k2*x3;
	if(x1==x2)
	{
		x=x1;
		y=k2*x+b2;
		if(k1==k2)
		y=k1*x+b1;
	}
	else if (x3==x4)
	{
		x=x3;
		y=k1*x+b1;
	}
	else 
	{
		x=(b2-b1)/(k1-k2);
		if(y1==y2)
			y=y1;
		else if(y3==y4)
			y=y3;
		else
		y=k1*x+b1;
	}
	if(x1>x2)
	{
		x1=bx.x;
		x2=ax.x;
	}
	if(y1>y2)
	{
		y1=bx.y;
		y2=ax.y;
	}
	if(x3>x4)
	{
		x3=dx.x;
		x4=cx.x;
		
	}
	if(y3>y4)
	{
		y3=dx.y;
		y4=cx.y;
	}
	if(((x1<=x&&x<=x2))&&((x3<=x&&x<=x4)))
	{	
		if(((y1<=y&&y<=y2))&&((y3<=y&&y<=y4)))return true;
		return false;
	}
	return false;

 }
  __host__ void drawLine(Point a,Point b)
 {
	glBegin(GL_LINES);
	glVertex2f(a.x,a.y);
	glVertex2f(b.x,b.y);
	glEnd();
 }
 __device__ __host__ bool isPointInCircle(Point *points,Point point)
 {

	 float a=points[0].y-points[1].y;
	 float b=points[1].x-points[0].x;
	 float c=points[2].x-points[0].x;
	 float d=points[2].y-points[0].y;

	 float x1,x2,x3;
	 float y1,y2,y3;
	 float cosin=((a*c+b*d)/(sqrt(a*a+b*b)*sqrt(c*c+d*d)));
	 float s;

	 if(cosin>0)
	 {
		 x1=points[1].x-point.x;
		 x2=points[0].x-point.x;
		 x3=points[2].x-point.x;

		 y1=points[1].y-point.y;
		 y2=points[0].y-point.y;
		 y3=points[2].y-point.y;
	 }
	 else if(cosin<0)
	 {
		 x1=points[0].x-point.x;
		 x2=points[1].x-point.x;
		 x3=points[2].x-point.x;

		 y1=points[0].y-point.y;
		 y2=points[1].y-point.y;
		 y3=points[2].y-point.y;
	 }
	 else
	 {
		return false;
	 }
	 s=(x1*x1 + y1*y1)*(y2*x3 - x2*y3) + (x2*x2 + y2*y2)*(x1*y3 - y1*x3) + (x3*x3 + y3*y3)*(y1*x2 - x1*y2) ;

	if( s <= 0)

		return true;
	return false;
 }
 __device__ __host__ void addLine(Line *lines,int *sizeL,Point a,Point b)
 {
	 lines[*sizeL].a=a;
	 lines[*sizeL].b=b;
	*sizeL=*sizeL+1;
	 
 }
 
 void connectWithFourthPoint(Point *points,Line *lines,int *sizeL)
 {
	float minDistance=sqrt((points[0].x-points[3].x)*(points[0].x-points[3].x)+(points[0].y-points[3].y)*(points[0].y-points[3].y));
	float distance1=sqrt((points[1].x-points[3].x)*(points[1].x-points[3].x)+(points[1].y-points[3].y)*(points[1].y-points[3].y));
	float distance2=sqrt((points[2].x-points[3].x)*(points[2].x-points[3].x)+(points[2].y-points[3].y)*(points[2].y-points[3].y));
	if(distance1<minDistance)
	{
		if(isIntersect(points[3],points[1],points[2],points[0])==false)
		{
		addLine(lines,sizeL,points[1],points[3]);
		}
		if(distance2<minDistance)
			{
				addLine(lines,sizeL,points[2],points[3]);
				if(isIntersect(points[3],points[0],points[2],points[1])==false)
				{
					addLine(lines,sizeL,points[3],points[0]);
				}
			}
		else if(distance2>=minDistance)
		{	
			addLine(lines,sizeL,points[0],points[3]);
			if(isIntersect(points[3],points[2],points[1],points[0])==false)
			{
				addLine(lines,sizeL,points[3],points[2]);
			}
		}
	}
	else if(distance1>=minDistance)
	{
		if(isIntersect(points[3],points[0],points[2],points[1])==false)
		{
		addLine(lines,sizeL,points[0],points[3]);
		}
		if(distance2<distance1)
		{
			addLine(lines,sizeL,points[2],points[3]);
			if(isIntersect(points[3],points[1],points[2],points[0])==false)
			{
				addLine(lines,sizeL,points[3],points[1]);
			}
		}
		else if(distance2>distance1)
		{
			addLine(lines,sizeL,points[1],points[3]);
			if(isIntersect(points[3],points[2],points[0],points[1])==false)
			{
				addLine(lines,sizeL,points[3],points[2]);
			}
		}
	}
 }
 __device__ __host__ bool isDelaunayCondition(Point *points,Point *mas,int length)
  {
	for(int i=0;i<length;++i)
	{
		if(isPointInCircle(points,mas[i])==false)
		return false;
	}
	return true;
  }
 
  __device__  float getAngle(Point Ax,Point Bx,Point Cx)
	{
		 float a=Ax.y-Bx.y;
		 float b=Ax.x-Bx.x;
		 float c=Cx.x-Bx.x;
		 float d=Cx.y-Bx.y;
		 if(sqrt(a*a+b*b)*sqrt(c*c+d*d)!=0)
		 return acos((a*d+b*c)/(sqrt(a*a+b*b)*sqrt(c*c+d*d)))*(180/3.14);
		 return 0.0;
	}
	__device__ void findBorderLine(int start1,int start2,int end1,int end2,short c,short dir,Point *mas1,Point *mas2,Point *SearchPoint1,Point *SearchPoint2,float Y1,float Y2)
{
	for(int i=start1;c*i<=c*end1;i+=c)
	{
		if(dir*mas1[i].x<dir*(*SearchPoint1).x)
		{
			Point tmp;
			tmp.x=(*SearchPoint1).x;
			tmp.y=Y1;
			if(!isIntersect(*SearchPoint2,mas1[i],*SearchPoint1,tmp))
			{
				*SearchPoint1=mas1[i];
			}
			else if((*SearchPoint1).x!=mas1[start1].x&&(*SearchPoint1).y!=mas1[start1].y)
			{
				break;
			}
			
		}
	}
	for(int i=start2;c*i<=c*end2;i+=c)
	{
		if(dir*mas2[i].x<dir*(*SearchPoint2).x)
		{		
			Point tmp;
			tmp.x=mas2[i].x;
			tmp.y=Y2;
			if(isIntersect(*SearchPoint2,*SearchPoint1,mas2[i],tmp))
			{
				*SearchPoint2=mas2[i];
				break;
			}
		}
	}


}

__device__ void buildTriangulation(Point *a,Point *b,int *sizeB,int *sizeA,Point *newA,Point *newB,Line *lines,int *sizeL,Line *newLines,int *sizeNL,Line downLine,int oldSizeA,int oldSizeB,bool *wasAddOldPoints)
 {

	 if((*newA).x!=downLine.a.x||(*newA).y!=downLine.a.y||(*newB).x!=downLine.b.x||(*newB).y!=downLine.b.y)
	 { 
		bool BA=false,AB=false;
		float min1=180.0;
		float min2=180.0;
	
		Point points[3];
		points[0]=(*newA);
		points[1]=(*newB);

		if((*newA).y==a[(*sizeA)-1].y&&(*newB).y==a[(*sizeA)-1].y)
		{
				(*sizeA)=(*sizeA)-1;
		}
		points[2]=a[(*sizeA)-1];
	
		BA=isDelaunayCondition(points, b,oldSizeB+1)&&isDelaunayCondition(points, a,oldSizeA+1);
		
		if((*newA).y==b[(*sizeB)-1].y&&(*newB).y==b[(*sizeB)-1].y)
		{
				(*sizeB)=(*sizeB)-1;
		}
		points[2]=b[(*sizeB)-1];
		AB=isDelaunayCondition(points, a,oldSizeA+1)&&isDelaunayCondition(points, b,oldSizeB+1);
	
		if(BA&&AB)
		{
			if(*wasAddOldPoints)
			{
				for(int i=0;i<*sizeNL;i++)
				{
					if((newLines[i].a.x== (*newA).x&&newLines[i].a.y== (*newA).y)&&( newLines[i].b.x==b[(*sizeB)-1].x&& newLines[i].b.y==b[(*sizeB)-1].y)||
						(newLines[i].a.x==b[(*sizeB)-1].x&&newLines[i].a.y== b[(*sizeB)-1].y)&&( newLines[i].b.x==(*newA).x&& newLines[i].b.y==(*newA).y))
						{
							AB=false;
							break;
						}
					if((newLines[i].a.x== (*newB).x&&newLines[i].a.y== (*newB).y)&&( newLines[i].b.x==a[(*sizeA)-1].x&& newLines[i].b.y==a[(*sizeA)-1].y)||
						(newLines[i].a.x==a[(*sizeA)-1].x&&newLines[i].a.y== a[(*sizeA)-1].y)&&( newLines[i].b.x==(*newB).x&& newLines[i].b.y==(*newB).y))
						{
							BA=false;
							break;
						}
				}
			}
			if(AB&&BA)
			{
				
				float alfa1=getAngle( (*newB), (*newA), a[(*sizeA)-1]);
				float alfa2=getAngle( (*newA), (*newB), b[(*sizeB)-1]);
				 min1=alfa1;
				 min2=alfa2;
				if(alfa1<180.0&&alfa2<180.0)
				{
 
					alfa1=getAngle( (*newA), (*newB), a[(*sizeA)-1]);
					if(180.0-(alfa1+min1)<min1)min1=180.0-(alfa1+min1);
					if(alfa1<min1)min1=alfa1;

 
					alfa2=getAngle( (*newB), (*newA), b[(*sizeB)-1]);
					if(180.0-(alfa2+min2)<min2)min2=180.0-(alfa2+min2);
					if(alfa2<min2)min2=alfa2; 
				}
				else if(alfa1>=180.0)min1=0;
				else if(alfa2>=180.0)min2=0;
			} 
		}
		if((min1>=min2&&min1!=180.0)||(BA&&!AB))
		{
			points[2]=a[(*sizeA)-1];
			if(*wasAddOldPoints)
			{
				for(int i=0;i<*sizeNL;i++)
				{
					
					if((newLines[i].a.x== (*newB).x&&newLines[i].a.y== (*newB).y)&&( newLines[i].b.x==a[(*sizeA)-1].x&& newLines[i].b.y==a[(*sizeA)-1].y)||
						(newLines[i].a.x==a[(*sizeA)-1].x&&newLines[i].a.y== a[(*sizeA)-1].y)&&( newLines[i].b.x==(*newB).x&& newLines[i].b.y==(*newB).y))
						{
							BA=false;
							(*sizeA)=(*sizeA)-1;
							break;
						}
				}
			}
			
	
			if(BA==true)
			{
				for(int i=0;i<*sizeL;i++)
				{
				if((lines[i].a.x!=(*newB).x||lines[i].a.y!= (*newB).y)&& (lines[i].b.x!=a[(*sizeA)-1].x|| lines[i].b.y!=a[(*sizeA)-1].y))
					if((lines[i].a.x!=a[(*sizeA)-1].x||lines[i].a.y!= a[(*sizeA)-1].y)&&( lines[i].b.x!=(*newB).x|| lines[i].b.y!=(*newB).y))
					if(isIntersect( lines[i].a, lines[i].b, (*newB), a[(*sizeA)-1]))
					{
						for (int j=i;j<*sizeL-1;j++)
						{
							lines[j]=lines[j+1];
						}
						*sizeL=*sizeL-1;
						i--;
					}
				}
				addLine(newLines,sizeNL,(*newB),a[(*sizeA)-1]);
				(*newA)=a[(*sizeA)-1];
				if((*sizeA)>1)
				{
					(*sizeA)=(*sizeA)-1;
				}
			}
			
		  
		}
		else if((min2>min1&&min2<180.0)||(!BA&&AB))
		{
			if(*wasAddOldPoints)
			{
				for(int i=0;i<*sizeNL;i++)
				{
					if((newLines[i].a.x== (*newA).x&&newLines[i].a.y== (*newA).y)&&( newLines[i].b.x==b[*sizeB-1].x&& newLines[i].b.y==b[*sizeB-1].y)||
						(newLines[i].a.x==b[*sizeB-1].x&&newLines[i].a.y== b[*sizeB-1].y)&&( newLines[i].b.x==(*newA).x&& newLines[i].b.y==(*newA).y))
						{
							AB=false;
							*sizeB=(*sizeB)-1;
							break;
						}
				}
			}
		
	
			if(AB==true)
			{
				for(int i=0;i<*sizeL;i++)
				{
				if((lines[i].a.x!= (*newA).x||lines[i].a.y!= (*newA).y)&&( lines[i].b.x!=b[*sizeB-1].x|| lines[i].b.y!=b[*sizeB-1].y))
					if((lines[i].a.x!=b[*sizeB-1].x||lines[i].a.y!= b[*sizeB-1].y)&&( lines[i].b.x!=(*newA).x|| lines[i].b.y!=(*newA).y))
					if(isIntersect( lines[i].a, lines[i].b, *newA, b[*sizeB-1]))
					{
						for (int j=i;j<*sizeL-1;j++)
						{
							lines[j]=lines[j+1];
						}
						*sizeL=*sizeL-1;
						i--;
					}
				}
				addLine(newLines,sizeNL,*newA,b[*sizeB-1]);
				*newB=b[*sizeB-1];
				if(*sizeB>1)
				{
					*sizeB=(*sizeB)-1;
				}
			}
		}
		else if(!BA&&!AB)
		{
			if(*sizeA>0&&a[*sizeA-1].y>(*newA).y&&a[*sizeA-1].y>(*newB).y)
			{
				*sizeA=*sizeA-1;
			}
			if(*sizeB>0&&b[*sizeB-1].y>(*newA).y&&b[*sizeB-1].y>(*newB).y)
			{
				*sizeB=*sizeB-1;
			}
		
			int end=0;
			Point tmp;
			if((*sizeA)>(*sizeB))end=(*sizeA);
			else end=(*sizeB);
			for (int i=0;i<end;i++)
			{
				if((*sizeA)-1-i>=0)
				{
					points[2]=a[(*sizeA)-1-i];
					if(isDelaunayCondition(points, a,oldSizeA+1)&&isDelaunayCondition(points, b,oldSizeB+1))
					{	tmp=a[(*sizeA)-1];
						a[(*sizeA)-1]=a[(*sizeA)-1-i];
						a[(*sizeA)-1-i]=tmp;
						break;
					}	
				}
				if((*sizeB)-1-i>=0)
				{
					points[2]=b[(*sizeB)-1-i];
					if(isDelaunayCondition(points, a,oldSizeA+1)&&isDelaunayCondition(points, b,oldSizeB+1))
					{
						tmp=b[(*sizeB)-1];
						b[(*sizeB)-1]=b[(*sizeB)-1-i];
						b[(*sizeB)-1-i]=tmp;
						break;
					}
				}
				if(i==end-1)
				{
						Point tmp;
						*wasAddOldPoints=true;
						for(int i=0;i<oldSizeA;i++)
						{
							if(a[oldSizeA-1-i].x==(*newA).x&&a[oldSizeA-1-i].y==(*newA).y)
							{
								tmp=a[oldSizeA-1-i];
								a[oldSizeA-1-i]=a[oldSizeA-1];
								a[oldSizeA-1]=tmp;
								(*sizeA)=oldSizeA+1;
								break;
							}	
						}
						for(int i=0;i<oldSizeB;i++)
						{
							if(b[oldSizeB-1-i].x==(*newB).x&&b[oldSizeB-1-i].y==(*newB).y)
							{
								tmp=b[oldSizeB-1-i];
								b[oldSizeB-1-i]=b[oldSizeB-1];
								b[oldSizeB-1]=tmp;
								(*sizeB)=oldSizeB+1;
								break;
							}
						}
				}
			}
		}
	 }
	 
 }
 
__device__ void makePolygonConvex(Point *a,Point *b,int *sizeA,int*sizeB,Line *newLines,int *sizeNL,Line *lines,int *sizeL)
 {
	Line topLine;
	Line downLine;
		
	quickSort(a,(*sizeA)-1,1,-1);
	quickSort(b, (*sizeB)-1,1,1);
	
	topLine.a=a[(*sizeA)-1];
	topLine.b=b[(*sizeB)-1];

	(*sizeA)=(*sizeA)-1;
	(*sizeB)=(*sizeB)-1;
	
	quickSort(a,(*sizeA)-1,1,1);
	quickSort(b, (*sizeB)-1,1,-1);
	
	
	downLine.a=a[0];
	downLine.b=b[0];
	
			
	if(topLine.a.y<topLine.b.y)
	{
		findBorderLine((*sizeA),(*sizeB),0,0,-1,1,a,b,&topLine.a,&topLine.b,downLine.a.y,downLine.b.y);
	}
	else if(topLine.a.y>topLine.b.y)
	{
	
		findBorderLine((*sizeB),(*sizeA),0,0,-1,-1,b,a,&topLine.b,&topLine.a,downLine.b.y,downLine.a.y);
	}
	for(int i=0;i<(*sizeA);i++)
	{
		if(a[(*sizeA)-1-i].x==topLine.a.x&&a[(*sizeA)-1-i].y==topLine.a.y)
		{
			Point tmp=a[(*sizeA)-1-i];
			a[(*sizeA)-1-i]=a[(*sizeA)];
			a[(*sizeA)]=tmp;
			break;
		}
	}
	for(int i=0;i<(*sizeB);i++)
	{
		if(b[(*sizeB)-1-i].x==topLine.b.x&&b[(*sizeB)-1-i].y==topLine.b.y)
		{
			Point  tmp=b[(*sizeB)-1-i];
			b[(*sizeB)-1-i]=b[(*sizeB)];
			b[(*sizeB)]=tmp;
			break;
		}
	}
				
	addLine(newLines,sizeNL,topLine.a,topLine.b);
	
	if(downLine.a.y>downLine.b.y)
	{
			findBorderLine(0,0,(*sizeA),(*sizeB),1,1,a,b,&downLine.a,&downLine.b,topLine.a.y,topLine.b.y);
			
	}
	else if(downLine.a.y<downLine.b.y)
	{
		findBorderLine(0,0,(*sizeB),(*sizeA),1,-1,b,a,&downLine.b,&downLine.a,topLine.b.y,topLine.a.y);
			
	}
	Point newA,newB;
	newA=topLine.a;
	newB=topLine.b;
	int oldsizeA=(*sizeA);
	int oldsizeB=(*sizeB);

	bool wasAddOldPoints=false;
	int c=0;
	while(newA.x!=downLine.a.x||newA.y!=downLine.a.y||newB.x!=downLine.b.x||newB.y!=downLine.b.y)
	{
	c++;
	buildTriangulation(a,b,sizeB,sizeA,&(newA),&(newB),lines,sizeL,newLines,sizeNL,downLine,oldsizeA,oldsizeB,&wasAddOldPoints);
	if(c>=50)break;
	}
 }


__global__ void connectTwoTriangulations(Point *points,int *counts,int sizeC,Line *lines,int *sizeL,int *countsSum,int *sizeLs,int *linesSum)
 {
	 int index=2*threadIdx.x;
	 int idx=threadIdx.x;
	 int sizeA=0;
		 int sizeB=0;
	
	
		__shared__ int cashe[SIZE/6+2];
		if(index+1<sizeC)
		{ 
			 sizeA=counts[index];
			 sizeB=counts[index+1];
		}
		 Line newLines[SIZE];
		int sizeNL=0;
		Point a[SIZE];
		Point b[SIZE]; 
		Line ls[4*SIZE];

		int end=sizeA;
		if(sizeB>sizeA&&sizeB>sizeLs[idx])
			end=sizeB;
		else if(sizeLs[idx]>sizeA&&sizeLs[idx]>sizeB)
			end=sizeLs[idx];
		for(int i=0;i<end;i++)
		{
			if(i<sizeA)
			{
				if(index-1>=0)
					a[i]=points[i+countsSum[index-1]];
				else 
					a[i]=points[i];
			}
			if(i<sizeB)
				b[i]=points[i+countsSum[index]];
			if(i<sizeLs[idx])
			{
				if(idx-1>=0)
					ls[i]=lines[i+linesSum[idx-1]];
				else 
					ls[i]=lines[i];
			}
				
		}
	
		if(index+1<sizeC)
		{
			makePolygonConvex(a,b,&sizeA,&sizeB,newLines,&sizeNL,ls,&sizeLs[idx]);//,indexes[idx],&sizeI[idx]
		}
		cashe[idx]=sizeNL+sizeLs[idx];
		__syncthreads();

		int s=0;
		for(int j=0;j<idx;j++)
		{
			s+=cashe[j];
			
		}

		for(int i=0;i<cashe[idx];i++)
		{
			if(i<sizeLs[idx])
				lines[s+i]=ls[i];
			else
				lines[s+i]=newLines[i-sizeLs[idx]];

		}
		sizeLs[idx]=cashe[idx];
		if(idx==blockDim.x-1)
		{

			*sizeL=s+sizeLs[idx];
		}

		linesSum[idx]=s+sizeLs[idx];
		__syncthreads();

		if(idx==0)
		{
			for(int i=0;i<blockDim.x;i++)
			{
				counts[i]=counts[2*i]+counts[2*i+1];
				countsSum[i]=countsSum[2*i+1];
				sizeLs[i]=sizeLs[2*i]+sizeLs[2*i+1];
				linesSum[i]=linesSum[2*i+1];
			}
			linesSum[blockDim.x]=linesSum[blockDim.x-1];
			sizeLs[blockDim.x]=0;
			counts[blockDim.x]=0;
		}
 } 
void divideIntoTriangles(int n,Point *points,Line *lines,int *sizeL,int *counts,int *sizeC,int *sizeLs);
 void splitPoints(int size1,int size2,Point *points,Line *lines,int *sizeL,int *counts,int *sizeC,int *sizeLs)
 {
	Point *p1= (Point *) malloc(size1*sizeof(Point));
	Point *p2= (Point *) malloc(size2*sizeof(Point));
	int size=size2;
	if(size1>size2)
	{
		size=size1;
	}
	 for(int i=0;i<size;++i)
	 {
			if(i<size1)
			{
				p1[i]=points[i];
			}
			if(i<size2)
			{
				p2[i]=points[i+size1];
			}
	 }
		 
	divideIntoTriangles(size1,p1,lines,sizeL,counts,sizeC,sizeLs);
	divideIntoTriangles(size2,p2,lines,sizeL,counts,sizeC,sizeLs);

	free(p1);
	free(p2);
	 
 }
  void divideIntoTriangles(int n,Point *points,Line *lines,int *sizeL,int *counts,int *sizeC,int *sizeLs)
 {
	 if(n==3)
	 { 
		counts[*sizeC]=n;
		sizeLs[*sizeC]=n;
		*sizeC=*sizeC+1;
		
		 addLine(lines,sizeL,points[0],points[1]);
		 addLine(lines,sizeL,points[1],points[2]);
		 addLine(lines,sizeL,points[2],points[0]);

	 }
	 else if(n==4)
	 {
		counts[*sizeC]=n;
		sizeLs[*sizeC]=*sizeL;
		if(isPointInCircle(points,points[3])==true)
		{
			 addLine(lines,sizeL,points[0],points[1]);
			 addLine(lines,sizeL,points[1],points[2]);
			 addLine(lines,sizeL,points[2],points[0]);
			 connectWithFourthPoint(points,lines,sizeL); 
		}
		else
		{
			for(int i=0;i<3;++i)
			{
				addLine(lines,sizeL,points[3],points[i]);
				if(i+1==3)
				{
					if(isIntersect(points[i],points[0],points[3],points[i-1])==false)
					{
						addLine(lines,sizeL,points[i],points[0]);
					}
				}
				else if(i-1<0)
				{
					if(isIntersect(points[i],points[i+1],points[3],points[2])==false)
					{
						addLine(lines,sizeL,points[i],points[i+1]);
					}
				}
				else
				{
					if(isIntersect(points[i],points[i+1],points[3],points[i-1])==false)
					{
						addLine(lines,sizeL,points[i],points[i+1]);
					}
				}
			}
		}
		 sizeLs[*sizeC]=*sizeL-sizeLs[*sizeC];
		*sizeC=*sizeC+1;

	 }
	 else if(n==8)
	 {
		splitPoints(4,4,points,lines,sizeL,counts,sizeC,sizeLs);
		 
	 }
	 else if(n<12&&n>2)
	 {
		 int t=n-3;
		 splitPoints(3,t,points,lines,sizeL,counts,sizeC,sizeLs);
		}
	 else if(n>=12)
	 {
		int t=n/2;
		splitPoints(t,n-t,points,lines,sizeL,counts,sizeC,sizeLs);
	 }
	 else if(n==2)
	 {
		counts[*sizeC]=n;
		sizeLs[*sizeC]=1;
		*sizeC=*sizeC+1;
		 addLine(lines,sizeL,points[0],points[1]);
	 }
	
 }
void mainFunction(void) 
{
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	int size=SIZE;
	float mas[SIZE][2]={{0.0,0.0},{-0.2,0.1},{0.32,-0.15},{0.17,-0.29},{-0.41,0.01},{-0.12,-0.2},{-0.48,-0.1},{0.54,0.29},{0.2,0.38},{-0.52,-0.09},
       {0.46,0.12},{0.49,-0.34},{-0.37,-0.02},{-0.09,0.05},{-0.22,0.2},{-0.55,-0.05},{0.02,-0.24},{0.3,-0.2},{-0.32,-0.04},{0.47,0.26},
       {-0.03,0.25},{-0.1,0.2},{0.16,0.4},{-0.16,0.15},{-0.55,0.15},{0.54,-0.01},{-0.28,-0.19},{0.28,-0.26},{0.48,0.02},{-0.46,0.01},
       {0.31,-0.06},{0.1,-0.21},{-0.41,-0.34},{0.0,-0.08},{0.21,-0.22},{-0.17,0.11},{-0.37,0.23},{-0.48,0.16},{0.45,0.38},{-0.39,0.32},
       {0.03,0.12},{0.29,0.24},{-0.36,0.41},{-0.54,0.19},{0.27,0.33},{-0.46,-0.33},{-0.11,-0.23},{-0.41,0.23},{-0.395,0.04},{0.34,0.15}};
	Point *points;
	Line *lines= (Line *) malloc(4*size*sizeof(Line));
	int *counts=(int *) malloc((size/3+2)*sizeof(int));
	int *countsSum=(int *) malloc((size/3+2)*sizeof(int));
	int *linesSum=(int *) malloc((size/3+2)*sizeof(int));
	int sizeL=0;
	int sizeC=0;
	points= (Point *) malloc(size*sizeof(Point));
	 int *sizeLs= (int *) malloc((size/3+2)*sizeof(int));

	  for (int i = 0; i < size; ++i)
	  {
		 points[i].x = mas[i][0];
		 points[i].y=mas[i][1];
	  }
	  quickSort(points, size-1,0,0);
	
	divideIntoTriangles(size, points,lines,&sizeL,counts,&sizeC,sizeLs);
	if(size>4)
	{
	counts[sizeC]=0;
	sizeLs[sizeC]=0;
	countsSum[0]=counts[0];
	linesSum[0]=sizeLs[0];
	for(int i=0;i<=sizeC;i++)
	{
		if(i-1>=0)
		{
			countsSum[i]=counts[i]+countsSum[i-1];
			linesSum[i]=sizeLs[i]+linesSum[i-1];
			if(i%2!=0)
				linesSum[(i-1)/2]=linesSum[i];

		}
		if(2*i+1<=sizeC)
			sizeLs[i]=sizeLs[2*i]+sizeLs[2*i+1];
	}
	
	Point *dev_points;
	int *dev_counts,*dev_sizeL,*dev_countsSum,*dev_sizeLs,*dev_linesSum;
	Line *dev_lines;
	hipMalloc((void**)&dev_points,size*sizeof(Point));
	hipMalloc((void**)&dev_counts,(size/3+2)*sizeof(int));
	hipMalloc((void**)&dev_lines,4*size*sizeof(Line));
	hipMalloc((void**)&dev_sizeL,sizeof(int));
	hipMalloc((void**)&dev_countsSum,(size/3+2)*sizeof(int));
	hipMalloc((void**)&dev_sizeLs,(size/3+2)*sizeof(int));
	hipMalloc((void**)&dev_linesSum,(size/3+2)*sizeof(int));

	hipMemcpy(dev_points,points,size*sizeof(Point),hipMemcpyHostToDevice);
	hipMemcpy(dev_lines,lines,4*size*sizeof(Line),hipMemcpyHostToDevice);
	hipMemcpy(dev_sizeL,&sizeL,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_counts,counts,(size/3+2)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_sizeLs,sizeLs,(size/3+2)*sizeof(int),hipMemcpyHostToDevice);
	
	hipMemcpy(dev_countsSum,countsSum,(size/3+2)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_linesSum,linesSum,(size/3+2)*sizeof(int),hipMemcpyHostToDevice);
	int threads=0;
		while(sizeC!=1)
		{
			threads=sizeC/2;
			if(sizeC%2!=0)
				threads++;
			connectTwoTriangulations<<<1,threads>>>(dev_points, dev_counts, sizeC,dev_lines,dev_sizeL,dev_countsSum,dev_sizeLs,dev_linesSum);
			sizeC=sizeC-sizeC/2;
		}
	
hipMemcpy(&sizeL,dev_sizeL,sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(lines,dev_lines,sizeL*sizeof(Line),hipMemcpyDeviceToHost);
	
	hipFree(dev_points);
	hipFree(dev_counts);
	hipFree(dev_lines);
	hipFree(dev_sizeL);
	hipFree(dev_sizeLs);
	hipFree(dev_countsSum);
	hipFree(dev_linesSum);
	
	}
hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time,start,stop);
	printf("time: %f ms\n",time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
printf("%d\n",sizeL);
	
for(int i=0;i<sizeL;i++)
{
	drawLine(lines[i].a,lines[i].b);
}
    glutSwapBuffers();
	free(lines);
	free(counts);
	free(points);
	free(countsSum);
	free(linesSum);
	free(sizeLs);
	
}
int main(int argc, char **argv)
{
	
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowPosition(100,100);
	glutInitWindowSize(1000,800);
	glutCreateWindow("Triangulation");
	glutDisplayFunc(mainFunction);
	glutMainLoop();
	
	return 0;
}
